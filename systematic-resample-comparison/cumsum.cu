
#include <hip/hip_runtime.h>
#include <float.h>

/**
 * Exclusive prefix sum implementation
 * Requires that the length of the data is a power of 2 and
 * the number of threads in a block is also a power of 2.
 * Otherwise it will access illegal memory.
 */
__global__ void block_scan(double *input, double *output, double *block_sums, int n, int save_block_sums) {
    extern __shared__ double shared_mem[];
    const int bx = blockIdx.x * blockDim.x;
    const int tx = threadIdx.x;
    const int px = bx + tx;
    int offset = 1;

    // copy everything to shared memory
    // every thread copies two values
    shared_mem[2*tx]   = input[2*px];
    shared_mem[2*tx+1] = input[2*px+1];

    ////
    // up sweep
    ////
    for (int d = n >> 1; d > 0; d >>= 1) {
        __syncthreads();

        if (tx < d) {
            int ai = offset * (2*tx+1) - 1;
            int bi = offset * (2*tx+2) - 1;

            shared_mem[bi] += shared_mem[ai];
        }
        offset <<= 1;
    }

    if (tx == 0) {
        if (save_block_sums == 1) {
            // save block sum if we run in multiple blocks
            block_sums[blockIdx.x] = shared_mem[n-1];
        }
        // clear last element
        shared_mem[n-1] = 0;
    }

    ////
    // down sweep
    ////
    for (int d = 1; d < n; d <<= 1) {
        offset >>= 1;
        __syncthreads();

        if (tx < d) {
            int ai = offset * (2*tx+1) - 1;
            int bi = offset * (2*tx+2) - 1;

            // swap
            double t = shared_mem[ai];
            shared_mem[ai]  = shared_mem[bi];
            shared_mem[bi] += t;
        }
    }
    __syncthreads();

    // save scan result
    output[2*px]   = shared_mem[2*tx];
    output[2*px+1] = shared_mem[2*tx+1];
}

// __global__ void block_scan_32(float *input, float *output, float *block_sums, int n, int save_block_sums) {
//     extern __shared__ float shared_mem[];
//     const int bx = blockIdx.x * blockDim.x;
//     const int tx = threadIdx.x;
//     const int px = bx + tx;
//     int offset = 1;

//     // copy everything to shared memory
//     // every thread copies two values
//     shared_mem[2*tx]   = input[2*px];
//     shared_mem[2*tx+1] = input[2*px+1];

//     ////
//     // up sweep
//     ////
//     for (int d = n >> 1; d > 0; d >>= 1) {
//         __syncthreads();

//         if (tx < d) {
//             int ai = offset * (2*tx+1) - 1;
//             int bi = offset * (2*tx+2) - 1;

//             shared_mem[bi] += shared_mem[ai];
//         }
//         offset <<= 1;
//     }

//     if (tx == 0) {
//         if (save_block_sums == 1) {
//             // save block sum if we run in multiple blocks
//             block_sums[blockIdx.x] = shared_mem[n-1];
//         }
//         // clear last element
//         shared_mem[n-1] = 0;
//     }

//     ////
//     // down sweep
//     ////
//     for (int d = 1; d < n; d <<= 1) {
//         offset >>= 1;
//         __syncthreads();

//         if (tx < d) {
//             int ai = offset * (2*tx+1) - 1;
//             int bi = offset * (2*tx+2) - 1;

//             // swap
//             float t = shared_mem[ai];
//             shared_mem[ai]  = shared_mem[bi];
//             shared_mem[bi] += t;
//         }
//     }
//     __syncthreads();

//     // save scan result
//     output[2*px]   = shared_mem[2*tx];
//     output[2*px+1] = shared_mem[2*tx+1];
// }


__global__ void add_partial_sums(double *output, double *block_sums) {
    const size_t bx = 2 * blockDim.x * blockIdx.x;
    const size_t tx = threadIdx.x;
    double block_sum = block_sums[blockIdx.x];

    output[bx + 2*tx]   += block_sum;
    output[bx + 2*tx+1] += block_sum;
}

// __global__ void add_partial_sums_32(float *output, float *block_sums) {
//     const size_t bx = 2 * blockDim.x * blockIdx.x;
//     const size_t tx = threadIdx.x;
//     float block_sum = block_sums[blockIdx.x];

//     output[bx + 2*tx]   += block_sum;
//     output[bx + 2*tx+1] += block_sum;
// }


// __global__ void block_psum_mine(
//     float *g_in, float *g_out, int n)
// {
//     extern __shared__ float smem[];
//     const int tx = threadIdx.x;
//     int offset = 1;

//     int old_n = n;

    
//     if(old_n % 2 == 0) {
//         if(tx < old_n/2) {
//             smem[2*tx]   = g_in[2*tx];
//             smem[2*tx+1] = g_in[2*tx+1];
//         }
//     } else {
//         if(tx < old_n/2) {
//             smem[2*tx]   = g_in[2*tx];
//             smem[2*tx+1] = g_in[2*tx+1];
//         } else if(tx == old_n/2) {
//             smem[2*tx]   = g_in[2*tx];
//             smem[2*tx+1]   = 0;
//         }
//     }

//     if (tx == 0) {
//         for(int i = 0; i < old_n; i++) {
//             printf("%d: %f\,", i, smem[i]);
//         }
//         printf("\n");
//     }

//     // init

//     if(n % 2 != 0) {
//         n++;
//     }

//     ////
//     // up sweep
//     ////
//     for (int d = n >> 1; d > 0; d >>= 1) {
//         __syncthreads();

//         if (tx < d) {
//             int ai = offset * (2*tx+1) - 1;
//             int bi = offset * (2*tx+2) - 1;

//             printf("%d: %d, %d\n", d, ai, bi);

//             smem[bi] += smem[ai];
//         }
//         offset <<= 1;
//     }

//     // save block sum and clear last element
//     if (tx == 0) {
//         for(int i = 0; i < n; i++) {
//             printf("%d: %f\,", i, smem[i]);
//         }
//         printf("\n");
//         smem[n-1] = 0;
//     }

//     ////
//     // down sweep
//     ////
//     for (int d = 1; d < n; d <<= 1) {
//         offset >>= 1;
//         __syncthreads();

//         if (tx < d) {
//             int ai = offset * (2*tx+1) - 1;
//             int bi = offset * (2*tx+2) - 1;

//             printf("%d> %d, %d\n", offset, tx);
//             printf("%d: %d, %d\n", d, ai, bi);

//             // swap
//             printf("%d swapping %d %d\n", d, ai, bi);
//             printf("%d swapping %d %d\n", d, offset, tx);
//             float t = smem[ai];
//             smem[ai]  = smem[bi];
//             smem[bi] += t;
//         }

//         printf("%d here\n", d);
//     }
//     __syncthreads();

//     if(old_n % 2 == 0) {
//         if(tx < old_n/2) {
//             g_out[2*tx]   = smem[2*tx];
//             g_out[2*tx+1] = smem[2*tx+1];
//         }
//     } else {
//         if(tx < old_n/2) {
//             g_out[2*tx]   = smem[2*tx];
//             g_out[2*tx+1] = smem[2*tx+1];
//         } else if(tx == n/2) {
//             g_out[2*tx]   = smem[2*tx];
//         }
//     }
// }