
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdbool.h>
#include <math.h>
#include <hiprand/hiprand_kernel.h>

#define THREADS 1048576

extern "C" {
// __device__ curandState_t* states[THREADS];
__device__ double r[THREADS];
__device__ hiprandState_t *state;

/**
 * Parallel Systematic Resample
 * Requires cumsum to be an exclusive cumsum i.e. start with 0.
 *
 * Uses double precision to prevent imprecision of floats for higher values of n.
 */
__global__ void systematic(double *weights, double *cumsum, int n, int threads, double rand, int *indices) {
    int block_id = blockIdx.x + blockIdx.y * gridDim.x;
    int idx = block_id * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;

    int left;
    int right;
    for (int k = idx; k < n; k += threads) {
        left = (int)ceil(((cumsum[k]) * n) - rand);
        right = (int)ceil(((cumsum[k] + weights[k]) * n) - rand);

        for(int j = left; j < right; j++) {
            indices[j] = k;
        }
    }
}

__global__ void init_rng(int seed)
{

    state = new hiprandState_t;
    hiprand_init(seed, 0, 0, state);

    // int block_id = blockIdx.x + blockIdx.y * gridDim.x;
    // int idx = block_id * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;
    
    // // for (int k = idx; k < THREADS; k += 1024) {
    // curandState_t* s = new curandState_t;
    // curand_init(seed, idx, 0, s);
    // double u = (double)curand_uniform(s);
    // r[idx] = u;
    // }
}

__global__ void draw_uniform(int n, int threads)
{

    int block_id = blockIdx.x + blockIdx.y * gridDim.x;
    int idx = block_id * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;

    for (int k = idx; k < n; k += threads) {
        double u = (double)hiprand_uniform(state);
        r[k] = u;
    }

    
    // // for (int k = idx; k < THREADS; k += 1024) {
    // curandState_t* s = new curandState_t;
    // curand_init(seed, idx, 0, s);
    // double u = (double)curand_uniform(s);
    // r[idx] = u;
    // }
}

__global__ void stratified(double *weights, double *cumsum, int n, int threads, int *indices) {
    int block_id = blockIdx.x + blockIdx.y * gridDim.x;
    int idx = block_id * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;

    int left;
    int right;
    for (int k = idx; k < n; k += threads) {
        left = ceil(((cumsum[k]) * n) - r[(int)floor((cumsum[k]) * n)]);

        if(k == n-1) {
            right = n;
        } else {
            right = ceil(((cumsum[k] + weights[k]) * n) - r[(int)floor((cumsum[k] + weights[k]) * n)]);
        }

        for(int j = left; j < right; j++) {
            indices[j] = k;
        }
    }
}

}
